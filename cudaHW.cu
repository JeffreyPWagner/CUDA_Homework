// cudaHW.cu
//
// driver and kernel call


#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 256
 
__global__ void vDotProd_d (int *force, int *distance, int *result, int n)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int i = n / 2;
    if (x < n) {
        if (x < i) {
            force[x] = x + 1;
        } else if (x > i) {
            force[x] = x - ((x - i) * 2);
        } else {
            force[x] = x;
        }
        distance[x] = x % 10 + 1;
        result[x] = force[x] * distance[x];
    }	
}

extern "C" void gpuDotProd (int *result_h, int arraySize)
{
    int *force;
    int *distance;
    int *result;

	hipMalloc ((void**) &force, sizeof(int) * arraySize);
	hipMalloc ((void**) &distance, sizeof(int) * arraySize);
	hipMalloc ((void**) &result, sizeof(int) * arraySize);
	
	vDotProd_d <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (force, distance, result, arraySize);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipMemcpy (result_h, result, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	hipFree (force);
	hipFree (distance);
	hipFree (result);
}
