// cudaHW.cu
//
// driver and kernel call


#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 256
 
__global__ void vDotProd_d (int *force, int *distance, int *result, int n)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    // (x<n) ensures that only existing elements get processed in a block+
    int i = n / 2;
    if (x < n) {
        if (x < i) {
            force[x] = x + 1;
        } else if (x > i) {
            force[x] = x - ((x - i) * 2);
        } else {
            force[x] = x;
        }
        distance[x] = x % 10 + 1;
        result[x] = force[x] * distance[x];
    }	
}

extern "C" void gpuDotProd (int *result_h, int arraySize)
{
    int *force;
    int *distance;
    int *result;

	hipMalloc ((void**) &force, sizeof(int) * arraySize);
	hipMalloc ((void**) &distance, sizeof(int) * arraySize);
	hipMalloc ((void**) &result, sizeof(int) * arraySize);
	
	// (float) ensures we are not doing integer division
	// ceil() ensures all data elements get processed
	vDotProd_d <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (force, distance, result, arraySize);
	
	// tests that everything worked; if not, print legible error message
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipMemcpy (result_h, result, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	hipFree (force);
	hipFree (distance);
	hipFree (result);
}
